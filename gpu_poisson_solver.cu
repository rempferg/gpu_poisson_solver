#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hipfft/hipfft.h>

#define PI_FLOAT 3.14159265358979323846264338327f

#define OUTPUT
//#define OUTPUT_GF
//#define OUTPUT_CHARGE
//#define OUTPUT_CHARGE_FFT
//#define OUTPUT_CHARGE_FFT_GF
//#define OUTPUT_POTENTIAL

void displayDeviceProperties(hipDeviceProp_t* pDeviceProp);

__global__ void createGreensFunc(hipfftReal* greensfunc, unsigned int Nx, unsigned int Ny, unsigned int Nz, float h) {
    unsigned int tmp;
    unsigned int coord[3];
    
    for(int i = blockDim.x*blockIdx.x+threadIdx.x; i < Nz * Ny * (Nx/2+1); i += gridDim.x*blockDim.x) {
        coord[0] = i % (Nx/2+1);
        tmp = i / (Nx/2+1);
        coord[1] = tmp % Ny;
        coord[2] = tmp / Ny;
        
        /* Setting 0th fourier mode to 0.0 enforces charge neutrality (effectively
           adds homogeneous counter charge). This is necessary, since the equation
           otherwise has no solution in periodic boundaries (an infinite amount of
           charge would create an infinite potential). */
        if(i == 0)
            greensfunc[i] = 0.0f;
        else
            greensfunc[i] = -0.5f * h * h / (cos(2.0f*PI_FLOAT*coord[0]/(hipfftReal)Nx) + cos(2.0f*PI_FLOAT*coord[1]/(hipfftReal)Ny) + cos(2.0f*PI_FLOAT*coord[2]/(hipfftReal)Nz) - 3.0f);
    }
}

__global__ void multiplyGreensFunc(hipfftComplex* data, hipfftReal* greensfunc, unsigned int N) {
    for(int i = blockDim.x*blockIdx.x+threadIdx.x; i < N; i += gridDim.x*blockDim.x) {
        data[i].x *= greensfunc[i];
        data[i].y *= greensfunc[i];
    }
}

int main(int argc, char** argv) {
    /* usage message */
    if(!(argc == 2 && strcmp(argv[1], "info") == 0) && argc != 5) {
        printf("USAGE: %s Nx Ny Nz h\n       %s info\n\nCalculates the electrostatic potential of a hardcoded charge distribution on a 3D grid of size Nx*Ny*Nz with grid spacing h.\n", argv[0], argv[0]);
        return 1;
    }
    
    /* cuda info */
    if(strcmp(argv[1], "info") == 0) {
        hipDeviceProp_t deviceProp;
        int devCount = 0;

        hipGetDeviceCount(&devCount);
        printf("Number of devices: %d\n", devCount);
        
        for (int i = 0; i < devCount; ++i) {
            memset(&deviceProp, 0, sizeof(deviceProp));
            
            if(hipGetDeviceProperties(&deviceProp, i) == hipSuccess)
                displayDeviceProperties(&deviceProp);
            else
                printf("\n%s", hipGetErrorString(hipGetLastError()));
        }
        
        return 0;
    }
    
    unsigned int Nx = atoi(argv[1]);
    unsigned int Ny = atoi(argv[2]);
    unsigned int Nz = atoi(argv[3]);
    float h = atof(argv[4]);
    
    printf("Calculating electrostatic potential on a %d*%d*%d grid with spacing %f\n", Nx, Ny, Nz, h);
    
    /* timing */
    float time = 0.0, time_tmp;
    hipEvent_t start, stop;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    /* allocations */
#if defined(OUTPUT) || defined(OUTPUT_GF) || defined(OUTPUT_CHARGE) || defined(OUTPUT_CHARGE_FFT) || defined(OUTPUT_CHARGE_FFT_GF) || defined(OUTPUT_POTENTIAL)
    FILE* fp;
#endif

    hipfftHandle plan_fft;
    hipfftHandle plan_ifft;
    hipfftComplex* data_dev;
    hipfftComplex* data_host;
    hipfftReal* data_real_host;
    hipfftReal* greensfunc_dev;
    hipfftReal* greensfunc_host;
    
    hipMalloc((void**) &data_dev, sizeof(hipfftComplex)*Nz*Ny*(Nx/2+1));
    
    if(hipGetLastError() != hipSuccess) {
        fprintf(stderr, "ERROR: Failed to allocate\n");
        return 1;
    }
    
    hipMalloc((void**) &greensfunc_dev, sizeof(hipfftReal)*Nz*Ny*(Nx/2+1));
    
    if(hipGetLastError() != hipSuccess) {
        fprintf(stderr, "ERROR: Failed to allocate\n");
        return 1;
    }
    
    hipHostMalloc((void**) &data_host, sizeof(hipfftComplex)*Nz*Ny*(Nx/2+1));
    
    if(hipGetLastError() != hipSuccess) {
        fprintf(stderr, "ERROR: Failed to allocate\n");
        return 1;
    }
    
    data_real_host = (hipfftReal*) data_host;
    
    hipHostMalloc((void**) &greensfunc_host, sizeof(hipfftReal)*Nz*Ny*(Nx/2+1));
    
    if(hipGetLastError() != hipSuccess) {
        fprintf(stderr, "ERROR: Failed to allocate\n");
        return 1;
    }
    
    /* greens function */
    printf("Creating greens function in device memory\n");
    
    createGreensFunc<<<14,32*32>>>(greensfunc_dev, Nx, Ny, Nz, h);
    
#if defined(OUTPUT) || defined(OUTPUT_GF)
    printf("Output of greens function: gf.vtk\n");
    
    hipMemcpy(greensfunc_host, greensfunc_dev, sizeof(hipfftReal)*Nz*Ny*(Nx/2+1), hipMemcpyDeviceToHost);
    
    if((fp = fopen("gf.vtk", "w")) == NULL) {
        fprintf(stderr, "ERROR: Could not open file\n");
        return 1;
    }
    
    fprintf(fp, "# vtk DataFile Version 2.0\ngreens_function\nASCII\n\nDATASET STRUCTURED_POINTS\nDIMENSIONS %u %u %u\nORIGIN 0 0 0\nSPACING %f %f %f\n\nPOINT_DATA %u\nSCALARS greens_function float 1\nLOOKUP_TABLE default\n", Nx, Ny, Nz, h, h, h, Nx*Ny*Nz);

    for(int z = 0; z < Nz; z++) {
        for(int y = 0; y < Ny; y++)
            for(int x = 0; x < Nx; x++)
                if(x >= Nx/2+1)
                    fprintf(fp, " %f", greensfunc_host[Ny*(Nx/2+1)*(Nz-z-1)+(Nx/2+1)*(Ny-y-1)+(Nx-x-1)]);
                else
                    fprintf(fp, " %f", greensfunc_host[Ny*(Nx/2+1)*z+(Nx/2+1)*y+x]);
            
        fprintf(fp, "\n");
    }

    fclose(fp);
#endif
    
    /* charge density */
    printf("Writing charge density in host memory\n");
    
    for(int z = 0; z < Nz; z++)
        for(int y = 0; y < Ny; y++)
            for(int x = 0; x < Nx; x++)
                if((x-Nx/2)*(x-Nx/2) + (y-Ny/2)*(y-Ny/2) + (z-Nz/2)*(z-Nz/2) <= 5*5/(h*h)) //homogeneously chargeed sphere of radius 5
                    data_real_host[Ny*Nx*z+Nx*y+x] = h*h*h;
                else
                    data_real_host[Ny*Nx*z+Nx*y+x] = 0.0;
                
    printf("Copying charge density to device\n");
    
    hipMemcpy(data_dev, data_host, sizeof(hipfftComplex)*Nz*Ny*(Nx/2+1), hipMemcpyHostToDevice);
    
    if(hipGetLastError() != hipSuccess) {
        fprintf(stderr, "ERROR: Failed to copy\n");
        return 1;
    }
    
#if defined(OUTPUT) || defined(OUTPUT_CHARGE)
    printf("Output of charge density: charge.vtk\n");
    
    if((fp = fopen("charge.vtk", "w")) == NULL) {
        fprintf(stderr, "ERROR: Could not open file\n");
        return 1;
    }
    
    fprintf(fp, "# vtk DataFile Version 2.0\ncharge_density\nASCII\n\nDATASET STRUCTURED_POINTS\nDIMENSIONS %u %u %u\nORIGIN 0 0 0\nSPACING %f %f %f\n\nPOINT_DATA %u\nSCALARS charge_density float 1\nLOOKUP_TABLE default\n", Nx, Ny, Nz, h, h, h, Nx*Ny*Nz);

    for(int z = 0; z < Nz; z++) {
        for(int y = 0; y < Ny; y++)
            for(int x = 0; x < Nx; x++)
                fprintf(fp, " %f", data_real_host[Ny*Nx*z+Nx*y+x]);
        
        fprintf(fp, "\n");
    }

    fclose(fp);
#endif

    /* create 3D FFT plans */
    printf("Setting up FFT and iFFT plans\n");
    
    /* Notice how the directions x and z are exchanged. This is because for R2C
       transforms, cuda only stores half the results in the 3rd direction. At
       the same time cuda expects the fastest running index to be the one with
       only half the values stored, which effectively forces one to make the 3rd
       index (usually z) the fastest running one. I find this rather uncommon
       and want x to be the festest running index and z the slowest running, so
       I chose to exchange the two in the fourier transforms. */
    if(hipfftPlan3d(&plan_fft, Nz, Ny, Nx, HIPFFT_R2C) != HIPFFT_SUCCESS) {
        fprintf(stderr, "ERROR: Unable to create fft plan\n");
        return 1;
    }
    
    if(cufftSetCompatibilityMode(plan_fft, CUFFT_COMPATIBILITY_NATIVE) != HIPFFT_SUCCESS) {
        fprintf(stderr, "ERROR: Unable to set fft compatibility mode to native\n");
        return 1;
    }
        
    if(hipfftPlan3d(&plan_ifft, Nz, Ny, Nx, HIPFFT_C2R) != HIPFFT_SUCCESS) {
        fprintf(stderr, "ERROR: Unable to create ifft plan\n");
        return 1;
    }
    
    if(cufftSetCompatibilityMode(plan_ifft, CUFFT_COMPATIBILITY_NATIVE) != HIPFFT_SUCCESS) {
        fprintf(stderr, "ERROR: Unable to set ifft compatibility mode to native\n");
        return 1;
    }
    
    /* FFT in place */
    printf("Executing FFT in place\n");
    
    hipEventRecord(start, 0);
    
    if(hipfftExecR2C(plan_fft, (hipfftReal*) data_dev, data_dev) != HIPFFT_SUCCESS) {
        fprintf(stderr, "ERROR: Unable to execute FFT plan\n");
        return 1;
    }
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_tmp, start, stop);
    printf("Execution time: %f ms\n", time_tmp);
    time += time_tmp;
    
    if(hipDeviceSynchronize() != hipSuccess) {
        fprintf(stderr, "ERROR: Failed to synchronize\n");
        return 1;
    }
    
#if defined(OUTPUT) || defined(OUTPUT_CHARGE_FFT)
    /* retrieving result from device */
    printf("Retrieving result from device\n");
    
    hipMemcpy(data_host, data_dev, sizeof(hipfftComplex)*Nz*Ny*(Nx/2+1), hipMemcpyDeviceToHost);
    
    /* output result */
    printf("Output of FFT(charge_density): charge_fft.vtk\n");
    
    if((fp = fopen("charge_fft.vtk", "w")) == NULL) {
        fprintf(stderr, "ERROR: Could not open file\n");
        return 1;
    }
    
    fprintf(fp, "# vtk DataFile Version 2.0\ncharge_fft\nASCII\n\nDATASET STRUCTURED_POINTS\nDIMENSIONS %u %u %u\nORIGIN 0 0 0\nSPACING %f %f %f\n\nPOINT_DATA %u\nSCALARS charge_fft float 2\nLOOKUP_TABLE default\n", Nx, Ny, Nz, h, h, h, Nx*Ny*Nz);
    
    for(int z = 0; z < Nz; z++) {
        for(int y = 0; y < Ny; y++)
            for(int x = 0; x < Nx; x++)
                if(x >= Nx/2+1)
                    fprintf(fp, " %f %f", data_host[Ny*(Nx/2+1)*(Nz-z-1)+(Nx/2+1)*(Ny-y-1)+(Nx-x-1)].x/sqrt(Nx*Ny*Nz), -data_host[Ny*(Nx/2+1)*(Nz-z-1)+(Nx/2+1)*(Ny-y-1)+(Nx-x-1)].y/sqrt(Nx*Ny*Nz));
                else
                    fprintf(fp, " %f %f", data_host[Ny*(Nx/2+1)*z+(Nx/2+1)*y+x].x/sqrt(Nx*Ny*Nz), data_host[Ny*(Nx/2+1)*z+(Nx/2+1)*y+x].y/sqrt(Nx*Ny*Nz));
        
        fprintf(fp, "\n");
    }
    
    fclose(fp);
#endif
    
    /* multiplying with greens function */
    printf("Executing multiplication with greens function in place\n");
    
    hipEventRecord(start, 0);
    
    multiplyGreensFunc<<<14,32*32>>>(data_dev, greensfunc_dev, Nz*Ny*(Nx/2+1)); //18-fold occupation seems to be optimal for the GT520 and 32-fold for the C2050
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_tmp, start, stop);
    printf("Execution time: %f ms\n", time_tmp);
    time += time_tmp;
    
    if(hipDeviceSynchronize() != hipSuccess) {
        fprintf(stderr, "ERROR: Failed to synchronize\n");
        return 1;
    }
    
#if defined(OUTPUT) || defined(OUTPUT_CHARGE_FFT_GF)
    /* retrieving result from device */
    printf("Retrieving result from device\n");
    
    hipMemcpy(data_host, data_dev, sizeof(hipfftComplex)*Nz*Ny*(Nx/2+1), hipMemcpyDeviceToHost);
    
    /* output result */
    printf("Output of FFT(charge_density)*greensfunc: charge_fft_gf.vtk\n");
    
    if((fp = fopen("charge_fft_gf.vtk", "w")) == NULL) {
        fprintf(stderr, "ERROR: Could not open output file\n");
        return 1;
    }
    
    fprintf(fp, "# vtk DataFile Version 2.0\ncharge_fft_gf\nASCII\n\nDATASET STRUCTURED_POINTS\nDIMENSIONS %u %u %u\nORIGIN 0 0 0\nSPACING %f %f %f\n\nPOINT_DATA %u\nSCALARS charge_fft_gf float 2\nLOOKUP_TABLE default\n", Nx, Ny, Nz, h, h, h, Nx*Ny*Nz);
    
    for(int z = 0; z < Nz; z++) {
        for(int y = 0; y < Ny; y++)
            for(int x = 0; x < Nx; x++)
                if(x >= Nx/2+1)
                    fprintf(fp, " %f %f", data_host[Ny*(Nx/2+1)*(Nz-z-1)+(Nx/2+1)*(Ny-y-1)+(Nx-x-1)].x/sqrt(Nx*Ny*Nz), -data_host[Ny*(Nx/2+1)*(Nz-z-1)+(Nx/2+1)*(Ny-y-1)+(Nx-x-1)].y/sqrt(Nx*Ny*Nz));
                else
                    fprintf(fp, " %f %f", data_host[Ny*(Nx/2+1)*z+(Nx/2+1)*y+x].x/sqrt(Nx*Ny*Nz), data_host[Ny*(Nx/2+1)*z+(Nx/2+1)*y+x].y/sqrt(Nx*Ny*Nz));
        
        fprintf(fp, "\n");
    }
    
    fclose(fp);
#endif
    
    /* inverse FFT in place */
    printf("Executing iFFT in place\n");
    
    hipEventRecord(start, 0);
    
    if(hipfftExecC2R(plan_ifft, data_dev, (hipfftReal*) data_dev) != HIPFFT_SUCCESS) {
        fprintf(stderr, "ERROR: Unable to execute iFFT plan\n");
        return 1;
    }
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_tmp, start, stop);
    printf("Execution time: %f ms\n", time_tmp);
    time += time_tmp;
    
    if(hipDeviceSynchronize() != hipSuccess) {
        fprintf(stderr, "ERROR: Failed to synchronize\n");
        return 1;
    }
    
#if defined(OUTPUT) || defined(OUTPUT_POTENTIAL)
    /* retrieving result from device */
    printf("Retrieving result from device\n");
    hipMemcpy(data_host, data_dev, sizeof(hipfftComplex)*Nz*Ny*(Nx/2+1), hipMemcpyDeviceToHost);
    
    /* output result */
    printf("Output of iFFT(FFT(charge_density)*greensfunc): charge_fft_gf_ifft.vtk\n");
    
    if((fp = fopen("charge_fft_gf_ifft.vtk", "w")) == NULL) {
        fprintf(stderr, "ERROR: Could not open output file\n");
        return 1;
    }
    
    fprintf(fp, "# vtk DataFile Version 2.0\npotential\nASCII\n\nDATASET STRUCTURED_POINTS\nDIMENSIONS %u %u %u\nORIGIN 0 0 0\nSPACING %f %f %f\n\nPOINT_DATA %u\nSCALARS potential float 1\nLOOKUP_TABLE default\n", Nx, Ny, Nz, h, h, h, Nx*Ny*Nz);

    for(int z = 0; z < Nz; z++) {
        for(int y = 0; y < Ny; y++)
            for(int x = 0; x < Nx; x++)
                fprintf(fp, " %f", data_real_host[Ny*Nx*z+Nx*y+x]/(Nx*Ny*Nz));
        
        fprintf(fp, "\n");
    }

    fclose(fp);
#endif

    /* cleanup */
    printf("Cleanup\n");
    
    hipfftDestroy(plan_fft);
    hipfftDestroy(plan_ifft);
    
    hipFree(data_dev);
    hipFree(greensfunc_dev);
    hipFree(data_host);
    hipFree(greensfunc_host);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    printf("Net device execution time: %f ms\n", time);
    
    return 0;
}

void displayDeviceProperties(hipDeviceProp_t* pDeviceProp)
{
    if(!pDeviceProp)
        return;

    printf("\nDevice Name \t – %s ", pDeviceProp->name);
    printf("\n**************************************");
    printf("\nTotal Global Memory\t\t -%d KB", (int) pDeviceProp->totalGlobalMem/1024);
    printf("\nShared memory available per block \t – %d KB", (int) pDeviceProp->sharedMemPerBlock/1024);
    printf("\nNumber of registers per thread block \t – %d", pDeviceProp->regsPerBlock);
    printf("\nWarp size in threads \t – %d", pDeviceProp->warpSize);
    printf("\nMemory Pitch \t – %d bytes", (int) pDeviceProp->memPitch);
    printf("\nMaximum threads per block \t – %d", pDeviceProp->maxThreadsPerBlock);
    printf("\nMaximum Thread Dimension (block) \t – %d %d %d", pDeviceProp->maxThreadsDim[0], pDeviceProp->maxThreadsDim[1], pDeviceProp->maxThreadsDim[2]);
    printf("\nMaximum Thread Dimension (grid) \t – %d %d %d", pDeviceProp->maxGridSize[0], pDeviceProp->maxGridSize[1], pDeviceProp->maxGridSize[2]);
    printf("\nTotal constant memory \t – %d bytes", (int) pDeviceProp->totalConstMem);
    printf("\nCUDA ver \t – %d.%d", pDeviceProp->major, pDeviceProp->minor);
    printf("\nClock rate \t – %d KHz", pDeviceProp->clockRate);
    printf("\nTexture Alignment \t – %d bytes", (int) pDeviceProp->textureAlignment);
    printf("\nDevice Overlap \t – %s", pDeviceProp-> deviceOverlap?"Allowed":"Not Allowed");
    printf("\nNumber of Multi processors \t – %d\n", pDeviceProp->multiProcessorCount);
}
